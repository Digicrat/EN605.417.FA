
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>
#include "getopt.h"
#include <stdint.h>


#define ARRAY_SIZE 2560  // Value chosen to match # cores in a GTX 1080 GPU
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

#define BLOCK_SIZE 32 // Default value. Original example was 16.

/* TODO: Define more realistic limits. For now,
 * num_threads>ARRAY_SIZE/num_blocks is good enough to make these
 * limits redundant */
#define MAX_BLOCKS ARRAY_SIZE
#define MAX_THREADS ARRAY_SIZE

/* Declare  statically two arrays of ARRAY_SIZE each */
unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_data[ARRAY_SIZE];

/* Image Generation Configuration
 *  We'll use a simple RGB color scheme
 *  This can be extended to other schemes (ie: sRGB, IAB) if needed later
 */
#if 0 // 10-bit color space -- good in theory, but harder to display in a useful format
#define TWO_BYTE_COLOR
#define MAX_COLOR 1023
#define R_SHIFT 0
#define G_SHIFT 10
#define B_SHIFT 20
#define R_MASK 0x000003FF
#define G_MASK 0x000FFC00
#define B_MASK 0x3FF00000
#define S_MASK 0xC0000000 // reserved (ie: alpha channel)
#else
// Note: We still reserve 10-bits per channel, but only use 8 when outputting
#define MAX_COLOR 255
#define R_SHIFT 0
#define G_SHIFT 10
#define B_SHIFT 20
#define R_MASK 0x000003FF
#define G_MASK 0x000FFC00
#define B_MASK 0x3FF00000
#define S_MASK 0xC0000000 // reserved (ie: alpha channel)

#endif

#define GET_R(data) (data & R_MASK)
#define GET_G(data) ((data & G_MASK) >> G_SHIFT)
#define GET_B(data) ((data & B_MASK) >> B_SHIFT)

#define GET_Rxy(x,y) (GET_R(cpu_data[x*width+y]))
#define GET_Gxy(x,y) (GET_G(cpu_data[x*width+y]))
#define GET_Bxy(x,y) (GET_B(cpu_data[x*width+y]))


// Write cpu_data as a PPM-formatted image (http://netpbm.sourceforge.net/doc/ppm.html)
void write_image(unsigned int width, unsigned int height)
{
  char fn[64];
  FILE *f;
  #ifdef TWO_BYTE_COLOR
  uint16_t c[3];
  #else
  uint8_t c[3];
  #endif

  sprintf(fn, "%d-%d.ppm", width, height);
  f = fopen(fn, "wb");
  fprintf(f, "P6\n%i %i %i\n", width, height, MAX_COLOR);
  for (int y=0; y<height; y++) {
    for (int x=0; x<width; x++) {
      c[0] = GET_Rxy(x,y);
      c[1] = GET_Gxy(x,y);
      c[2] = GET_Bxy(x,y);
#ifdef TWO_BYTE_COLOR
      fwrite(c, 2, 3, f);
#else
      fwrite(c, 1, 3, f);
#endif
      //printf("%d,%d = %d %d %d\n", x,y,c[0],c[1],c[2]);
    }
  }
  fclose(f);
}

__global__
void what_is_my_id(unsigned int * block, unsigned int * thread, unsigned int * data)
{
  // blockNum * thradsPerBlock + threadNum
  const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  block[thread_idx] = blockIdx.x;
  thread[thread_idx] = threadIdx.x;

  data[thread_idx] =
    (threadIdx.x & R_MASK) | // threadIdx lower-bits sets the red color
    ((blockIdx.x*10<<G_SHIFT) & G_MASK) | // blockIdx lower-bits sets the green color
    ((thread_idx<<B_SHIFT) & B_MASK); // thread_idx lower-bits itself will be the blue value
	
}

void main_sub0(unsigned int num_threads, unsigned int num_blocks, int verbose)
{

	/* Declare pointers for GPU based params */
	unsigned int *gpu_block;
	unsigned int *gpu_thread;
	unsigned int *gpu_data;

	hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_data, ARRAY_SIZE_IN_BYTES);

	hipMemcpy( gpu_block, cpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );
	hipMemcpy( gpu_thread, cpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );
	hipMemcpy( gpu_data, gpu_data, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );

	/* Execute our kernel */
	what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_data);

	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy( cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost );
	hipMemcpy( cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost );
	hipMemcpy( cpu_data, gpu_data, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost );
	hipFree(gpu_block);
	hipFree(gpu_thread);
	hipFree(gpu_data);

	/* Iterate through the arrays and output */
	if (verbose) {
	  for(unsigned int i = 0; i < ARRAY_SIZE; i++)
	  {
	    printf("Thread: %2u - Block: %2u - Data: %08x - %03u %03u %03u\n",
		   cpu_thread[i],cpu_block[i],cpu_data[i],
		   GET_R(cpu_data[i]), GET_G(cpu_data[i]), GET_B(cpu_data[i]));
	  }
	}

	write_image(num_blocks, num_threads);
}

int main(int argc, char* argv[])
{
  unsigned int blk_size = BLOCK_SIZE;
  unsigned int num_threads = ARRAY_SIZE/blk_size;
  int c;
  int verbose = 0;
  
  while((c = getopt(argc, argv, "hvb:t:")) != -1) {
    switch(c) {
    case 'b':
      blk_size = atoi(optarg);
      break;
    case 't':
      num_threads = atoi(optarg);
      break;
    case 'v':
      verbose = 1;
      break;
    case 'h':
      printf("Usage: \n");
      printf("\t-h     Show this message\n");
      printf("\t-v     Enable verbose output mode.\n");
      printf("\t-b 32  Specify number of blocks to use (ie: 32 in this example). Default is %d\n", BLOCK_SIZE);
      printf("\t-t 32  Specify number of threads per block to use (ie: 32 in this example). Default is %d\n", ARRAY_SIZE/BLOCK_SIZE);
      return -1;
    default:
      printf("ERROR: Option %s is not supported, type h for usage info.\n", c);
      return -1;
    }
  }
  if (blk_size > MAX_BLOCKS)
  {
    printf("ERROR: blk_size (%d specified) must be <= max BLOCK_SIZE of %d\n", blk_size, BLOCK_SIZE);
    return -1;
  } else {
    printf("blk_size set to %d\n", blk_size);
  }
  if (num_threads > MAX_THREADS || num_threads > ARRAY_SIZE/blk_size)
  {
    printf("ERROR: num_threads (%d) cannot exceed %d, or %d/blk_size=%d\n", num_threads, MAX_THREADS, ARRAY_SIZE, ARRAY_SIZE/blk_size);
    return -1;
  } else {
    printf("num_threads set to %d\n", num_threads);
  }

  
  main_sub0(num_threads, blk_size, verbose);
	
  return EXIT_SUCCESS;
}
